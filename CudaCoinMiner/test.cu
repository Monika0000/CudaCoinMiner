#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "unit_test.h"

__global__ void findKernel(unsigned int* gpu_result) {
    unsigned int i = threadIdx.x;
    if (i == 500)
        *gpu_result = i;
}

int main2() {
    unsigned int result = 0;

    hipSetDevice(0);

    unsigned int* dev_result = NULL;
    hipMalloc((void**)&dev_result, sizeof(unsigned int));

    findKernel<<<1, 1000>>>(dev_result);
    hipDeviceSynchronize();

    hipMemcpy(&result, dev_result, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("%i", result);
    hipFree(dev_result); 

    getchar();

    return 1;
}
