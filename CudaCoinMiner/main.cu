#include <stdio.h>
// #include "unit_test.h"
#include <WinSock2.h>
#include "miner.cu"
#include "sha1.cu"

int main() {
    hipSetDevice(0);

    /*if (!check_sha1() || !check_sha1_2() || !check_sha1_3() || !check_sha1_4()) {
        printf("SHA1 is not working!\n");
        return -1;
    }
    else
        printf("Checking SHA1 is successful\n");*/

    SOCKET sock = connect_to_server("51.15.127.80", 2811);
    if (sock == INVALID_SOCKET)
        return -1;

    if (request_job(sock, 3)) {
        int result = process_job(sock);
        printf("Result: %i\n", result);
    }


    //hipSetDevice(0);
    return 0;
}

/*
unsigned int* dev_result = NULL;
hipMalloc((void**)&dev_result, sizeof(unsigned int));

char* dev_prefix = NULL;
hipMalloc((void**)dev_prefix, 41);
hipMemcpy(dev_prefix, prefix, 41, hipMemcpyHostToDevice);

char* dev_target = NULL;
hipMalloc((void**)dev_target, 41);
hipMemcpy(dev_target, job, 41, hipMemcpyHostToDevice);

unsigned int* dev_diff = NULL;
hipMalloc((void**)dev_diff, sizeof(unsigned int));
hipMemcpy(dev_diff, &diff, sizeof(unsigned int), hipMemcpyHostToDevice);

sha1Kernel<<<1000, 256>>>(dev_result, dev_prefix, dev_target, dev_diff);
hipDeviceSynchronize();

hipMemcpy(&result, dev_result, sizeof(unsigned int), hipMemcpyDeviceToHost);


hipFree(dev_result);
*/