#include <stdio.h>
// #include "unit_test.h"
#include <WinSock2.h>
#include "miner.cu"

DWORD WINAPI run_miner() {
    hipSetDevice(0);

    /*if (!check_sha1() || !check_sha1_2() || !check_sha1_3() || !check_sha1_4()) {
        printf("SHA1 is not working!\n");
        return -1;
    }
    else
        printf("Checking SHA1 is successful\n");*/

    SOCKET sock = connect_to_server("51.15.127.80", 2811);
    // SOCKET sock = connect_to_server("51.195.65.23", 9999);

    if (sock == INVALID_SOCKET)
        return -1;

    unsigned int result = 0;

    unsigned int* dev_result = NULL;
    hipMalloc((void**)&dev_result, sizeof(unsigned int));
    hipError_t cudaerror = hipGetLastError();
    if (cudaerror != hipSuccess) {
        printf("dev_result malloc error: %s\n", hipGetErrorString(cudaerror));
    }
    
    char* dev_prefix = NULL;
    hipMalloc((void**)&dev_prefix, 41);

    byte* dev_target = NULL;
    hipMalloc((void**)&dev_target, 20);

    unsigned int* dev_diff = NULL;
    hipMalloc((void**)&dev_diff, sizeof(unsigned int));

    while (true) {
        if (request_job(sock, 3)) {
            result = process_job(sock, dev_result, dev_prefix, dev_target, dev_diff);
            send_job(sock, result);
        }
        else
            break;
    }

    hipFree(dev_result);
    hipFree(dev_prefix);
    hipFree(dev_target);
    hipFree(dev_diff);

    return 0;
}

int main() {
    hipSetDevice(1);

    for (int i = 0; i < 4; i++) {
        CreateThread(NULL, 0, (LPTHREAD_START_ROUTINE)&run_miner, NULL, 0, NULL);
    }

    getchar();

    return 0;
}

/*
unsigned int* dev_result = NULL;
hipMalloc((void**)&dev_result, sizeof(unsigned int));

char* dev_prefix = NULL;
hipMalloc((void**)dev_prefix, 41);
hipMemcpy(dev_prefix, prefix, 41, hipMemcpyHostToDevice);

char* dev_target = NULL;
hipMalloc((void**)dev_target, 41);
hipMemcpy(dev_target, job, 41, hipMemcpyHostToDevice);

unsigned int* dev_diff = NULL;
hipMalloc((void**)dev_diff, sizeof(unsigned int));
hipMemcpy(dev_diff, &diff, sizeof(unsigned int), hipMemcpyHostToDevice);

sha1Kernel<<<1000, 256>>>(dev_result, dev_prefix, dev_target, dev_diff);
hipDeviceSynchronize();

hipMemcpy(&result, dev_result, sizeof(unsigned int), hipMemcpyDeviceToHost);


hipFree(dev_result);
*/